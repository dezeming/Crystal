#include "hip/hip_runtime.h"
/*
    Copyright (C) <2023>  <Dezeming>  <feimos@mail.ustc.edu.cn>

    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or any
    later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <https://www.gnu.org/licenses/>.

    Github site: <https://github.com/dezeming/Crystal>
*/


#include "Texture.h"

namespace CrystalAlgrithm {

// ConstantFloatTexture
ConstantFloatTexture::ConstantFloatTexture(float v) {
    val = v;
}
ConstantFloatTexture:: ~ConstantFloatTexture() {

}


// ConstantRGBATexture
ConstantRGBATexture::ConstantRGBATexture(float4 v) {
    val = v;
}
ConstantRGBATexture::~ConstantRGBATexture() {

}


// MapFloatTexture
MapFloatTexture::MapFloatTexture() {

}
MapFloatTexture::~MapFloatTexture() {

}


// MapRGBATexture
MapRGBATexture::MapRGBATexture() {

}
MapRGBATexture::~MapRGBATexture() {


}


// TextureManager
TextureManager::TextureManager() {

}
TextureManager::~TextureManager() {

}


}
