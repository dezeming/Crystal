/*
	Copyright (C) <2023>  <Dezeming>  <feimos@mail.ustc.edu.cn>

	This program is free software: you can redistribute it and/or modify
	it under the terms of the GNU General Public License as published by
	the Free Software Foundation, either version 3 of the License, or any
	later version.

	This program is distributed in the hope that it will be useful,
	but WITHOUT ANY WARRANTY; without even the implied warranty of
	MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
	GNU General Public License for more details.

	You should have received a copy of the GNU General Public License
	along with this program.  If not, see <https://www.gnu.org/licenses/>.

	Github site: <https://github.com/dezeming/Crystal>
*/

#include "cuda_common.cuh"
#include "cpu_Common.hpp"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <string>
#include <iostream>
#include <vector>

namespace CrystalAlgrithm {

void printCudaDevice() {
	COUT_BLUE("Cuda Support Devices") << std::endl;

	hipDeviceProp_t prop;
	int count;
	bool accuracy = Get_CUDA_ERROR(hipGetDeviceCount(&count));
	for (int i = 0; i < count; i++) {
		accuracy = Get_CUDA_ERROR(hipGetDeviceProperties(&prop, i));
		if (accuracy) {
			std::cout << "GPU sequence " + std::to_string(i) << std::endl;
			std::cout << "  GPU name " << prop.name << std::endl;
			std::cout << "  Compute capability " << prop.major << "." << prop.minor << std::endl;
			std::cout << "  Clock rate " << prop.clockRate << std::endl;
			std::cout << "  Total global memory " << prop.totalGlobalMem / 1024 / 1024 / 1024 << "GB" << std::endl;
			std::cout << "  Total constant memory " << prop.totalConstMem / 1024 << "KB" << std::endl;
			std::cout << "  Multiprocessor count " << prop.multiProcessorCount << std::endl;
		}

	}
}


void getCudaDevice(std::vector<CudaDeviceProperty> &cudadevices) {
	cudadevices.clear();
	hipDeviceProp_t prop;
	int count;
	bool accuracy = Get_CUDA_ERROR(hipGetDeviceCount(&count));
	for (int i = 0; i < count; i++) {
		accuracy = Get_CUDA_ERROR(hipGetDeviceProperties(&prop, i));
		if (accuracy) {

			CudaDeviceProperty cudadevice;
			cudadevice.GPU_name = prop.name;
			cudadevice.Compute_capability = std::to_string(prop.major) + "." + std::to_string(prop.minor);
			cudadevice.Clock_rate = std::to_string(prop.clockRate);
			cudadevice.Total_global_memory = std::to_string(prop.totalGlobalMem / 1024 / 1024 / 1024) + "GB";
			cudadevice.Total_constant_memory = std::to_string(prop.totalConstMem / 1024) + "KB";
			cudadevice.Multiprocessor_count = std::to_string(prop.multiProcessorCount);

			cudadevices.push_back(cudadevice);
		}
	}
}

std::string getCudaErrorStr(const char* error, const char* file, int line) {
	return (std::string(error) + " in " + std::string(file) + " at line " + std::to_string(line));
	//exit(EXIT_FAILURE);
}

bool getCudaError(hipError_t err, const char* file, int line) {
	if (err != hipSuccess) {
		COUT_RED("Error: ") << getCudaErrorStr(hipGetErrorString(err), file, line) << std::endl;
		return false;
	}
	else {
		return true;
	}
}

bool getCudaError(hipError_t err) {
	if (err != hipSuccess) {
		COUT_RED("Error: ") << hipGetErrorString(err) << std::endl;
		return false;
	}
	else {
		return true;
	}
}




}















