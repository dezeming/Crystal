#include "hip/hip_runtime.h"
/*
    Copyright (C) <2023>  <Dezeming>  <feimos@mail.ustc.edu.cn>

    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or any
    later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <https://www.gnu.org/licenses/>.

    Github site: <https://github.com/dezeming/Crystal>
*/

#include "Camera.h"
#include "CrystalAlgrithm/Scene/ScenePreset.h"
#include "CrystalAlgrithm/Utility/Sampling.h"

#include <hip/hip_runtime.h>

#define CameraDebug true

namespace CrystalAlgrithm {

// CameraPreset
CameraPreset::CameraPreset() {
	reset();
}
void CameraPreset::reset() {
	CameraType = "perspective";
	fov = 45.0f;
	lookatFrom[0] = 0.0f; lookatFrom[1] = 0.0f; lookatFrom[2] = 5.0f;
	lookatTo[0] = 0.0f; lookatTo[1] = 0.0f; lookatTo[2] = 0.0f;
}
bool CameraPreset::isValid() {
	if (fov < 5.0f || fov > 170.f) return false;
	// Currently only supports perspective projection cameras
	if (CameraType != "perspective") return false;
	return true;
}
std::string CameraPreset::ToString() {
	return "CameraPreset: "
		"  CameraType:[" + CameraType + "]" +
		"  fov:[" + std::to_string(fov) + "]" +
		" LookAt:from[" + Point3fToString(lookatFrom) + "]" +
		"to[" + Point3fToString(lookatTo) + "]";
}

// FilmPreset
FilmPreset::FilmPreset() {
	width = 1024;
	height = 768;
}
void FilmPreset::reset() {
	width = 1024;
	height = 768;
}
bool FilmPreset::isValid() {
	// Currently, only resolutions up to 4000 * 4000 are supported.
	if (width > 4000 || height > 4000) {
		return false;
	}
	return true;
}
std::string FilmPreset::ToString() {
	return "FilmPreset: "
		"  width:[" + std::to_string(width) + "]"
		+ "  height:[" + std::to_string(height) + "]";
}

// Film

Film::Film() {
	if (CameraDebug && !DisablePrintInfo)
		Print_CPU_Info("Create Film Object");

	finalUChar4Buffer_host = nullptr;
	finalFloat4Buffer_host = nullptr;

	finalUChar4Buffer_device = nullptr;
	finalFloat4Buffer_device = nullptr;
}

void Film::init(const Point2i& res, unsigned int type, float maxSampleLuminance) {
	if (CameraDebug && !DisablePrintInfo)
		Print_CPU_Info("Init Film Object");

	resolution = res;

	finalUChar4Buffer_host = new uchar4[res.x * res.y];
	finalFloat4Buffer_host = new float4[res.x * res.y];

	hipMalloc(&finalUChar4Buffer_device, res.x * res.y * sizeof(uchar4));
	hipMalloc(&finalFloat4Buffer_device, res.x * res.y * sizeof(float4));
}

Film::~Film() {
	if (CameraDebug && !DisablePrintInfo)
		Print_CPU_Info("Destroy Film Object");

	if (finalUChar4Buffer_device)
		hipFree(finalUChar4Buffer_device);
	if (finalFloat4Buffer_device)
		hipFree(finalFloat4Buffer_device);

	if (finalUChar4Buffer_host)
		delete[] finalUChar4Buffer_host;
	if (finalFloat4Buffer_host)
		delete [] finalFloat4Buffer_host;
}


// Camera
Camera::Camera(unsigned int type):
type(type){

}
Camera::~Camera() {

}

void Camera::GenerateRay(const CameraSample& sample, CameraRay* ray) const {
	// Compute raster and camera sample positions
	Point3f pFilm = Point3f(sample.pFilm.x, sample.pFilm.y, 0);
	Point3f pCamera = Raster2Camera(pFilm);
	*ray = CameraRay(Point3f(0, 0, 0), Normalize(Vector3f(pCamera)));

	// Modify ray for depth of field
	if (lensRadius > 0) {
		// Sample point on lens
		Point2f pLens = lensRadius * ConcentricSampleDisk(sample.pLens);

		// Compute point on plane of focus
		float ft = focalDistance / ray->direction.z;
		Point3f pFocus = (*ray)(ft);

		// Update ray for effect of lens
		ray->origin = Point3f(pLens.x, pLens.y, 0);
		ray->direction = Normalize(pFocus - ray->origin);
	}
	ray->isInVolume = isInVolume;
	*ray = Camera2World(*ray);

}
void Camera::GenerateRayDifferential(const CameraSample& sample,
	CameraRayDifferential* rd) const {

}


}

